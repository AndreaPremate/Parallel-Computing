#include "hip/hip_runtime.h"
#include "my_lib.h"
#include <limits.h>


int* add_padding(int* original, int cols, int rows) {

    int padded_rows = rows + 2;
    int padded_cols = cols + 2;
    int size_with_padding = (padded_rows) * (padded_cols);

    int* padded_matrix = (int*)malloc(size_with_padding * sizeof(int));

    // Inizializza la matrice con padding a zero
    memset(padded_matrix, 0, size_with_padding * sizeof(int));

    // Copia i valori
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            padded_matrix[(i + 1) * (padded_cols) + (j + 1)] = original[i * cols + j];
        }
    }

    return padded_matrix;
}

int* remove_padding(int* padded_matrix, int padded_cols, int padded_rows) {

    int original_rows = padded_rows - 2;
    int original_cols = padded_cols - 2;

    int* original_matrix = (int*)malloc(original_rows * original_cols * sizeof(int));
 
    // Copia i valori
    for (int i = 0; i < original_rows; ++i) {
        for (int j = 0; j < original_cols; ++j) {
            original_matrix[i * original_cols + j] = padded_matrix[(i + 1) * padded_cols + (j + 1)];
        }
    }

    return original_matrix;
}



__global__ void InitLabels(int* Labels, int SIZEX) {
    int SIZEXPAD = SIZEX + 2;
    int id = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    int cy = id / SIZEX;
    int cx = id - cy * SIZEX;
    int aPos = (cy + 1) * SIZEXPAD + cx + 1;
    int l = Labels[aPos];
    l *= aPos;
    Labels[aPos] = l;
}

__global__ void Scanning(int* Labels, int* IsNotDone, int SIZEX, int SIZEY) {
    int SIZEYPAD = SIZEY + 2;
    int SIZEXPAD = SIZEX + 2;
    unsigned int id = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int cy = id / SIZEX;
    unsigned int cx = id % SIZEX;
    unsigned int aPos = (cy + 1) * SIZEXPAD + cx + 1;

    if (aPos < SIZEXPAD * SIZEYPAD) { 
        unsigned int l = Labels[aPos];
        if (l) {
            unsigned int lw = Labels[aPos - 1];
            unsigned int le = Labels[aPos + 1];
            unsigned int ls = Labels[aPos - SIZEX - 2];
            unsigned int ln = Labels[aPos + SIZEX + 2];
            unsigned int minl = INT_MAX;

            if (lw) minl = lw;
            if (le && le < minl) minl = le;
            if (ls && ls < minl) minl = ls;
            if (ln && ln < minl) minl = ln;

            if (minl < l) {
                unsigned int ll = Labels[l];
                Labels[l] = min(ll, minl);
                IsNotDone[0] = 1;
            }
        }
    }
}

__global__ void Analysis(int* Labels, int SIZEX, int SIZEY) {
    int SIZEYPAD = SIZEY + 2;
    int SIZEXPAD = SIZEX + 2;
    unsigned int id = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

    unsigned int cy = id / SIZEX;
    unsigned int cx = id % SIZEX;
    unsigned int aPos = (cy + 1) * SIZEXPAD + cx + 1;

    if (aPos < SIZEXPAD * SIZEYPAD) { 
        unsigned int label = Labels[aPos];
        if (label) {
            unsigned int r = Labels[label];
            while (r != label) {
                label = Labels[r];
                r = Labels[label];
            }
            Labels[aPos] = label;
        }
    }
}


#define CREATEIMG 1
int main(int argc, char **argv) {

    initializeRandomSeed();
    int w, h;
    int *h_img_no_padding = read_bmp("C:/Path/to/Image/image.bmp", &w, &h);
    int *h_img = add_padding(h_img_no_padding, w, h);      
    #if (CREATEIMG)
    createColorMappedBMP("color_mapped.bmp", h_img, w+2, h+2);
    #endif
    int img_size = (w+2)*(h+2);
    
    // START TIMER
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Allocazione della memoria sul device
    int* d_img;
    hipMalloc((void**)&d_img, img_size * sizeof(int));

    // Copia dei dati dall'host al device
    hipMemcpy(d_img, h_img, img_size * sizeof(int), hipMemcpyHostToDevice);

    dim3 threadsPerBlock(64);
    dim3 blocksPerGrid((w+2 + threadsPerBlock.x - 1) / threadsPerBlock.x,
                   (h+2 + threadsPerBlock.y - 1) / threadsPerBlock.y);

    // Allocazione e inizializzazione di IsNotDone
    int *d_isNotDone;
    int h_isNotDone = 1; // 1 per entrare nel ciclo
    hipMalloc((void**)&d_isNotDone, sizeof(int));
    hipMemcpy(d_isNotDone, &h_isNotDone, sizeof(int), hipMemcpyHostToDevice);

    // Inizializza labels
    InitLabels<<<blocksPerGrid, threadsPerBlock>>>(d_img, w);

    // Ciclo Scanning e Analysis
    do {
        h_isNotDone = 0; 
        hipMemcpy(d_isNotDone, &h_isNotDone, sizeof(int), hipMemcpyHostToDevice);

        Scanning<<<blocksPerGrid, threadsPerBlock>>>(d_img, d_isNotDone, w, h);
        Analysis<<<blocksPerGrid, threadsPerBlock>>>(d_img, w, h);

        hipMemcpy(&h_isNotDone, d_isNotDone, sizeof(int), hipMemcpyDeviceToHost);
    } while (h_isNotDone);

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // Calcolo e stampa del tempo trascorso
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Tempo di esecuzione: %f ms\n", milliseconds);

    // Pulizia eventi CUDA
    hipEventDestroy(start);
    hipEventDestroy(stop);
    
    // Copia dei risultati indietro sull'host
    hipMemcpy(h_img, d_img, img_size * sizeof(int), hipMemcpyDeviceToHost);
    h_img_no_padding = remove_padding(h_img, w+2, h+2);
    #if (CREATEIMG)
    createColorMappedBMP("color_mapped_final.bmp", h_img_no_padding, w, h);
    #endif

    // Libera memoria
    free(h_img);
    free(h_img_no_padding);
    freeHashTable();
    hipFree(d_img);

    return 0;
}